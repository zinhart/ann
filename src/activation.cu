#include "hip/hip_runtime.h"
#include "ann/activation.hh"
#include "concurrent_routines/concurrent_routines.hh"
#include "concurrent_routines/concurrent_routines_error.hh"
namespace zinhart
{
  // wrappers for host functions to use to call kernels here, the wrappers will calculate the block_parameters and the threads per block
  std::int32_t call_activation(const ACTIVATION_NAME activation_name, const ACTIVATION_TYPE activation_type, double * device_Wx_plus_b, std::uint32_t current_layer_size)
  {
	hipError_t error_id;
	hipDeviceProp_t properties;
	hipGetDeviceProperties(&properties, 0);
	dim3 block_launch;
	std::int32_t warp_size = properties.warpSize;
	std::int32_t threads_per_block = (current_layer_size + warp_size -1) / warp_size * warp_size;
	if(threads_per_block > 4 * warp_size)
	  threads_per_block = 4 * warp_size;
	block_launch.x = (current_layer_size + threads_per_block - 1) / threads_per_block;// number of blocks
	block_launch.y = 1;
	block_launch.z = 1;
	//std::cout<<"current_layer_size: "<<current_layer_size<<" threads_per_block: "<<threads_per_block<<" warp_size: "<<warp_size <<" block_launch.x: " <<block_launch.x<< " block_launch.y: " <<block_launch.y<< " block_launch.z: " <<block_launch.z<<"\n";
	//call kernel
	activation_kernel<<<block_launch, threads_per_block>>>(activation_name, activation_type, device_Wx_plus_b, current_layer_size);
	hipDeviceSynchronize();
  	error_id = hipGetLastError();
	if(error_id != hipSuccess)
	{
	  std::cerr<<"activation_kernel failed to launch with error: "<<hipGetErrorString(error_id)<<"\n";
	  return 1;
	}
	return 0;
  }

  // this method does not synchronize interally
  std::int32_t call_activation(const ACTIVATION_NAME activation_name, const ACTIVATION_TYPE activation_type, double * device_Wx_plus_b, std::uint32_t current_layer_size, const hipStream_t & stream)
  {
	hipError_t error_id;
	hipDeviceProp_t properties;
	hipGetDeviceProperties(&properties, 0);
	dim3 block_launch;
	std::int32_t warp_size = properties.warpSize;
	std::int32_t threads_per_block = (current_layer_size + warp_size -1) / warp_size * warp_size;
	if(threads_per_block > 4 * warp_size)
	  threads_per_block = 4 * warp_size;
	block_launch.x = (current_layer_size + threads_per_block - 1) / threads_per_block;// number of blocks
	block_launch.y = 1;
	block_launch.z = 1;
	//std::cout<<"current_layer_size: "<<current_layer_size<<" threads_per_block: "<<threads_per_block<<" warp_size: "<<warp_size <<" block_launch.x: " <<block_launch.x<< " block_launch.y: " <<block_launch.y<< " block_launch.z: " <<block_launch.z<<"\n";
	//call kernel
	activation_kernel<<<block_launch, threads_per_block,0, stream>>>(activation_name, activation_type, device_Wx_plus_b, current_layer_size);
	//hipDeviceSynchronize();
  	error_id = hipGetLastError();
	if(error_id != hipSuccess)
	{
	  std::cerr<<"activation_kernel failed to launch with error: "<<hipGetErrorString(error_id)<<"\n";
	  return 1;
	}
	return 0;
  }

 /* 
	std::int32_t call_activation(ACTIVATION_NAME activation_name, ACTIVATION_TYPE activation_type, double * device_Wx_plus_b, double coefficient, std::uint32_t layer_size)
  {
	hipError_t error_id;
	hipDeviceProp_t properties;
	hipGetDeviceProperties(&properties, 0);
	std::int32_t warp_size = properties.warpSize;
	std::int32_t threads_per_block = (layer_size + warp_size - 1) / (warp_size * warp_size);
    if(threads_per_block > 4 * warp_size)
	  threads_per_block = 4 * warp_size;	  
	dim3 block_launch;
	block_launch.x = (layer_size + threads_per_block - 1) / threads_per_block;
	block_launch.y = 1; 
	block_launch.z = 1;
	//call kernel
	activation_kernel_coeff<<<block_launch, threads_per_block>>>(activation_name, activation_type, device_Wx_plus_b, coefficient, layer_size);
	hipDeviceSynchronize();
	error_id = hipGetLastError();
	if(error_id != hipSuccess)
	{
	  std::cerr<<"activation_kernel_coeff failed to launch with error: "<<hipGetErrorString(error_id);
	  return 1;
	}
	//copy memory from host to device
	return 0;
  }*/


  //activation function kernels here
	__global__ void activation_kernel(ACTIVATION_NAME activation_name, ACTIVATION_TYPE activation_type, double * device_Wx_plus_b, std::uint32_t layer_size) //everything that's not leaky relu, elu, or softmax
  {

	const std::uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("thread_id: %d\n", thread_id);
	if(thread_id >= layer_size)
	  return;
	if(activation_name == ACTIVATION_NAME::IDENTITY)
	{
	  activation<identity> f;
	  device_Wx_plus_b[thread_id] =  f(device_Wx_plus_b[thread_id], activation_type);
	}
	else if(activation_name == ACTIVATION_NAME::SIGMOID)
	{
	  activation<sigmoid> f;
	  device_Wx_plus_b[thread_id] =  f(device_Wx_plus_b[thread_id], activation_type);
	}
	else if(activation_name == ACTIVATION_NAME::SOFTPLUS)
	{
	  activation<softplus> f;
	  device_Wx_plus_b[thread_id] =  f(device_Wx_plus_b[thread_id], activation_type);
	}
	else if(activation_name == ACTIVATION_NAME::TANH)
	{
	  activation<hyperbolic_tangent> f;
	  device_Wx_plus_b[thread_id] =  f(device_Wx_plus_b[thread_id], activation_type);
	}
	else if(activation_name == ACTIVATION_NAME::RELU)
	{
	  activation<relu> f;
	  device_Wx_plus_b[thread_id] =  f(device_Wx_plus_b[thread_id], activation_type);
	}
	else
	  return;
  }
  __global__ void activation_kernel_coeff(ACTIVATION_NAME activation_name, ACTIVATION_TYPE activation_type, double * device_Wx_plus_b, double coefficient, std::uint32_t layer_size)//leaky relu or elu
  {
	std::uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
	printf("thread_id: %d\n", thread_id);
	if(thread_id > layer_size)
	  return;
	switch(activation_name)
	{
	  case ACTIVATION_NAME::LEAKY_RELU:
		device_Wx_plus_b[thread_id] = activation_leaky_relu(activation_type, device_Wx_plus_b[thread_id], coefficient);
		break;
	  case ACTIVATION_NAME::EXP_LEAKY_RELU:
		device_Wx_plus_b[thread_id] = activation_exponential_leaky_relu(activation_type, device_Wx_plus_b[thread_id], coefficient);
		break;
	  default:
		return;
	}
  }
  __global__ void activation_kernel_softmax(ACTIVATION_TYPE activation_type, double * device_Wx_plus_b, std::uint32_t layer_size)
  {
	//to do
	std::uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
	if(thread_id > layer_size)
  	  return;
	return;
  }
}
